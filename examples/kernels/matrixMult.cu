
#include <hip/hip_runtime.h>
template <int TILE_WIDTH>
__global__ void MatrixMulty1unfolded(float *Md, float *Nd, float *Pd,
                                     int width) {
  const int row{4 * blockIdx.y * blockDim.y + threadIdx.y};
  const int col{blockIdx.x * blockDim.x + threadIdx.x};
  const int row_l{4 * threadIdx.y};
  const int col_l{threadIdx.x};
  __shared__ float Ml[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nl[TILE_WIDTH][TILE_WIDTH];

  float sum0{0.0f};
  float sum1{0.0f};
  float sum2{0.0f};
  float sum3{0.0f};
  for (int m{0}; m < width; m += TILE_WIDTH) {
    __syncthreads();
    Ml[row_l][col_l] = Md[(row)*width + m + col_l];
    Ml[row_l + 1][col_l] = Md[(row + 1) * width + m + col_l];
    Ml[row_l + 2][col_l] = Md[(row + 2) * width + m + col_l];
    Ml[row_l + 3][col_l] = Md[(row + 3) * width + m + col_l];

    Nl[row_l][col_l] = Nd[width * (m + row_l) + col];
    Nl[row_l + 1][col_l] = Nd[width * (m + 1 + row_l) + col];
    Nl[row_l + 2][col_l] = Nd[width * (m + 2 + row_l) + col];
    Nl[row_l + 3][col_l] = Nd[width * (m + 3 + row_l) + col];

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) {
      sum0 += Ml[row_l][k] * Nl[k][col_l];
      sum1 += Ml[row_l + 1][k] * Nl[k][col_l];
      sum2 += Ml[row_l + 2][k] * Nl[k][col_l];
      sum3 += Ml[row_l + 3][k] * Nl[k][col_l];
    }
    __syncthreads();
  }
  Pd[row * width + col] = sum0;
  Pd[(row + 1) * width + col] = sum1;
  Pd[(row + 2) * width + col] = sum2;
  Pd[(row + 3) * width + col] = sum3;
}

template <int TILE_WIDTH, int GRANULARITY>
__global__ void MatrixMulty1(float *Md, float *Nd, float *Pd, int width) {
  const int col{blockIdx.x * blockDim.x + threadIdx.x};
  const int row{GRANULARITY * blockIdx.y * blockDim.y + threadIdx.y};
  const int col_l{threadIdx.x};
  const int row_l{GRANULARITY * threadIdx.y};
  __shared__ float Ml[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nl[TILE_WIDTH][TILE_WIDTH];

  float sum[GRANULARITY];
#pragma unroll
  for (int n{0}; n < GRANULARITY; ++n) {
    sum[n] = 0.0f;
  }

  for (int m{0}; m < width; m += TILE_WIDTH) {
#pragma unroll
    for (int n{0}; n < GRANULARITY; ++n) {
      Ml[row_l + n][col_l] = Md[(row + n) * width + (m + col_l)];
    }
#pragma unroll
    for (int n{0}; n < GRANULARITY; ++n) {
      Nl[row_l + n][col_l] = Nd[(m + row_l + n) * width + col];
    }

    __syncthreads();
#pragma unroll
    for (int k{0}; k < TILE_WIDTH; ++k) {
#pragma unroll
      for (int n{0}; n < GRANULARITY; ++n) {
        sum[n] += Ml[row_l + n][k] * Nl[k][col_l];
      }
    }
    __syncthreads();
  }
#pragma unroll
  for (int n{0}; n < GRANULARITY; ++n) {
    Pd[(row + n) * width + col] = sum[n];
  }
}

// see:
// https://github.com/kberkay/Cuda-Matrix-Multiplication/blob/master/matrix_Multiplication.cu#L100
template <int BLOCK_SIZE>
__global__ void MatrixMulty2(float *left, float *right, float *res, int dim) {

  int i, j;
  float temp = 0;

  __shared__ float Left_shared_t[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Right_shared_t[BLOCK_SIZE][BLOCK_SIZE];

  // Row i of matrix left
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  for (int tileNUM = 0; tileNUM < gridDim.x; tileNUM++) {

    // Column j of matrix left
    j = tileNUM * BLOCK_SIZE + threadIdx.x;
    i = tileNUM * BLOCK_SIZE + threadIdx.y;
    // Load left[i][j] to shared mem

    Left_shared_t[threadIdx.y][threadIdx.x] =
        left[row * dim + j]; // Coalesced access
    // Load right[i][j] to shared mem

    Right_shared_t[threadIdx.y][threadIdx.x] =
        right[i * dim + col]; // Coalesced access
    // Synchronize before computation
    __syncthreads();

    // Accumulate one tile of res from tiles of left and right in shared mem
    for (int k = 0; k < BLOCK_SIZE; k++) {

      temp += Left_shared_t[threadIdx.y][k] *
              Right_shared_t[k][threadIdx.x]; // no shared memory bank conflict
    }
    // Synchronize
    __syncthreads();
  }
  // Store accumulated value to res
  res[row * dim + col] = temp;
}

extern "C" __global__ void MatrixMultyNaive(float *A, float *B, float *C,
                                            int N) {

  const int ROW = blockIdx.y * blockDim.y + threadIdx.y;
  const int COL = blockIdx.x * blockDim.x + threadIdx.x;

  float sum{0};

  if (ROW < N && COL < N) {
    for (auto i{0}; i < N; i++) {
      sum += A[ROW * N + i] * B[i * N + COL];
    }
  }
  C[ROW * N + COL] = sum;
}
