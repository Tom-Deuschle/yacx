
#include <hip/hip_runtime.h>
extern "C" __global__ void sumArrayOnGPU(float *A, float *B, float *C) {
  int i_inBlock = threadIdx.x + threadIdx.y * blockDim.x +
                  threadIdx.z * blockDim.y * blockDim.x;
  int blockID = blockIdx.x;
  int i = i_inBlock + blockID * (blockDim.x * blockDim.y * blockDim.z);
  C[i] = A[i] + B[i];
}